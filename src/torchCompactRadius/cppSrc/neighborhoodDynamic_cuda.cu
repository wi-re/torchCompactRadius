#include "hip/hip_runtime.h"
#include "neighborhood.h"
#include "neighborhoodDynamic.h"

template<std::size_t dim = 2, typename scalar_t = float>
__global__ void buildNeighborhoodCudaDispatcher(int32_t numParticles,
                                                cptr_t<int32_t, 1> neighborOffsets, ptr_t<int64_t, 1> neighborList_i, ptr_t<int64_t, 1> neighborList_j,
                                                cptr_t<scalar_t, 2> queryPositions, cptr_t<scalar_t, 1> querySupport, int32_t searchRange,
                                                cptr_t<scalar_t, 2> sortedPositions, cptr_t<scalar_t, 1> sortedSupport,
                                                cptr_t<int32_t, 2> hashTable, int32_t hashMapLength,
                                                cptr_t<int32_t, 2> cellTable, cptr_t<int32_t, 1> numCells,
                                                cptr_t<int32_t, 2> offsets, scalar_t hCell, cptr_t<scalar_t, 1> minDomain, cptr_t<scalar_t, 1> maxDomain, cptr_t<bool, 1> periodicity,
                                                supportMode searchMode) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        buildNeighborhood<dim, scalar_t>(i, neighborOffsets, neighborList_i, neighborList_j, queryPositions, querySupport, searchRange, sortedPositions, sortedSupport, hashTable, hashMapLength, cellTable, numCells, offsets, hCell, minDomain, maxDomain, periodicity, searchMode);
    }
}
template<std::size_t dim = 2, typename scalar_t = float>
__global__ void countNeighborsForParticleCudaDispatcher(int32_t numParticles,
                                                        ptr_t<int32_t, 1> neighborCounters,
                                                        cptr_t<scalar_t, 2> queryPositions, cptr_t<scalar_t, 1> querySupport, int32_t searchRange,
                                                        cptr_t<scalar_t, 2> sortedPositions, cptr_t<scalar_t, 1> sortedSupport,
                                                        cptr_t<int32_t, 2> hashTable, int32_t hashMapLength,
                                                        cptr_t<int32_t, 2> cellTable, cptr_t<int32_t, 1> numCellsVec,
                                                        cptr_t<int32_t, 2> offsets,
                                                        scalar_t hCell, cptr_t<scalar_t, 1> minDomain, cptr_t<scalar_t, 1> maxDomain, cptr_t<bool, 1> periodicity,
                                                        supportMode searchMode) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numParticles) {
        countNeighborsForParticle<dim, scalar_t>(i, neighborCounters, queryPositions, querySupport, searchRange, sortedPositions, sortedSupport, hashTable, hashMapLength, cellTable, numCellsVec, offsets, hCell, minDomain, maxDomain, periodicity, searchMode);
    }
}

// #ifdef CUUDA_VERSION
#ifdef CUDA_VERSION
#include <hip/hip_runtime.h>

void cuda_error_check() {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
    error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(error));
    }
}
#endif

template<typename Func, typename... Ts>
void launchKernel(Func kernel, int32_t numParticles, Ts&&... args) {
    int32_t blockSize;  // Number of threads per block
    int32_t minGridSize;  // Minimum number of blocks required for the kernel
    int32_t gridSize;  // Number of blocks to use

    // Compute the maximum potential block size for the kernel
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, kernel);
    // cuda_error_check();
    gridSize = (numParticles + blockSize - 1) / blockSize;

    kernel<<<gridSize, blockSize>>>(numParticles, std::forward<Ts>(args)...);
    // cuda_error_check();
}


void buildNeighborhoodCuda(torch::Tensor neighborOffsets, torch::Tensor neighborList_i, torch::Tensor neighborList_j,
    torch::Tensor queryPositions, torch::Tensor querySupport, int32_t searchRange,
    torch::Tensor sortedPositions, torch::Tensor sortedSupport,
    torch::Tensor hashTable, int32_t hashMapLength,
    torch::Tensor cellTable, torch::Tensor numCells,
    torch::Tensor offsets, float hCell, torch::Tensor minDomain, torch::Tensor maxDomain, torch::Tensor periodicity,
    supportMode searchMode) {
    int32_t numParticles = queryPositions.size(0);
    
    int32_t threads = 1024;
    int32_t blocks = (int32_t)floor(numParticles / threads) + (numParticles % threads == 0 ? 0 : 1);

#define args numParticles, \
neighborOffsets.packed_accessor32<int32_t,1, traits>(), neighborList_i.packed_accessor32<int64_t,1, traits>(), neighborList_j.packed_accessor32<int64_t,1, traits>(), \
queryPositions.packed_accessor32<scalar_t, 2, traits>(), querySupport.packed_accessor32<scalar_t,1, traits>(), searchRange, \
sortedPositions.packed_accessor32<scalar_t, 2, traits>(), sortedSupport.packed_accessor32<scalar_t,1, traits>(), \
hashTable.packed_accessor32<int32_t,2, traits>(), hashMapLength, \
cellTable.packed_accessor32<int32_t,2, traits>(), numCells.packed_accessor32<int32_t,1, traits>(), \
offsets.packed_accessor32<int32_t,2, traits>(), \
hCell, minDomain.packed_accessor32<scalar_t,1, traits>(), maxDomain.packed_accessor32<scalar_t,1, traits>(), periodicity.packed_accessor32<bool,1, traits>(), searchMode

    int32_t dim = queryPositions.size(1);
    if(dim == 1)
        AT_DISPATCH_FLOATING_TYPES(queryPositions.scalar_type(), "buildNeighborhoodCuda", [&] {
            launchKernel(buildNeighborhoodCudaDispatcher<1, scalar_t>, args);
        });
        // buildNeighborhoodCudaDispatcher<1><<<blocks, threads>>>(args);
    else if(dim == 2)
        AT_DISPATCH_FLOATING_TYPES(queryPositions.scalar_type(), "buildNeighborhoodCuda", [&] {
            launchKernel(buildNeighborhoodCudaDispatcher<2, scalar_t>, args);
        });
        // buildNeighborhoodCudaDispatcher<2><<<blocks, threads>>>(args);
    else if(dim == 3)
        AT_DISPATCH_FLOATING_TYPES(queryPositions.scalar_type(), "buildNeighborhoodCuda", [&] {
            launchKernel(buildNeighborhoodCudaDispatcher<3, scalar_t>, args);
        });
        // buildNeighborhoodCudaDispatcher<3><<<blocks, threads>>>(args);
    else throw std::runtime_error("Unsupported dimensionality");

    // cuda_error_check();

#undef args
}

void countNeighborsForParticleCuda(
    torch::Tensor neighborCounters, 
    torch::Tensor queryPositions, torch::Tensor querySupport, int32_t searchRange, 
    torch::Tensor sortedPositions, torch::Tensor sortedSupport,
    torch::Tensor hashTable, int32_t hashMapLength,
    torch::Tensor cellTable, torch::Tensor numCellsVec, 
    torch::Tensor offsets,
    float hCell, torch::Tensor minDomain, torch::Tensor maxDomain, torch::Tensor periodicity,
    supportMode searchMode) {
    int32_t numParticles = queryPositions.size(0);

#define args \
        numParticles, \
        neighborCounters.packed_accessor32<int32_t,1, traits>(), \
        queryPositions.packed_accessor32<scalar_t,2, traits>(), querySupport.packed_accessor32<scalar_t,1, traits>(), searchRange, \
        sortedPositions.packed_accessor32<scalar_t,2, traits>(), sortedSupport.packed_accessor32<scalar_t,1, traits>(), \
        hashTable.packed_accessor32<int32_t,2, traits>(), hashMapLength, \
        cellTable.packed_accessor32<int32_t,2, traits>(), numCellsVec.packed_accessor32<int32_t,1, traits>(), \
        offsets.packed_accessor32<int32_t,2, traits>(), \
        hCell, minDomain.packed_accessor32<scalar_t, 1, traits>(), maxDomain.packed_accessor32<scalar_t, 1, traits>(), periodicity.packed_accessor32<bool, 1, traits>(), searchMode

    int32_t dim = queryPositions.size(1);
    // std::cout << "dim: " << dim << std::endl;
    if (dim == 1)
    AT_DISPATCH_FLOATING_TYPES(queryPositions.scalar_type(), "countNeighborsForParticleCuda", [&] {
        launchKernel(countNeighborsForParticleCudaDispatcher<1, scalar_t>, args);
    });
        // countNeighborsForParticleCudaDispatcher<1><<<blocks, threads>>>(args);
    else if (dim == 2)
    AT_DISPATCH_FLOATING_TYPES(queryPositions.scalar_type(), "countNeighborsForParticleCuda", [&] {
        launchKernel(countNeighborsForParticleCudaDispatcher<2, scalar_t>, args);
    });
        // countNeighborsForParticleCudaDispatcher<2><<<blocks, threads>>>(args);
    else if (dim == 3)
    AT_DISPATCH_FLOATING_TYPES(queryPositions.scalar_type(), "countNeighborsForParticleCuda", [&] {
        launchKernel(countNeighborsForParticleCudaDispatcher<3, scalar_t>, args);
    });
        // countNeighborsForParticleCudaDispatcher<3><<<blocks, threads>>>(args);
    else throw std::runtime_error("Unsupported dimensionality");

#undef args
}
