#include <multiLevelMemory/buildNeighborhoodDense.h>
#include <multiLevelMemory/mlmUtil.h>

void buildNeighborhoodDense_cuda(buildNeighborhoodDense_functionArguments_t) {
    int32_t nQuery = queryPositions_.size(0);
    auto scalar = queryPositions_.scalar_type();
    auto dim = queryPositions_.size(1);

    auto wrappedArguments = std::make_tuple(queryPositions_.is_cuda(), buildNeighborhoodDense_arguments_t_);

    DISPATCH_FUNCTION_DIM_SCALAR(dim, scalar, "buildNeighborsMLMParticle", [&]() {
        auto functionArguments = std::apply(buildNeighborhoodDense_getFunctionArguments<scalar_t>, wrappedArguments);
        launchKernel([] __device__(auto... args) { buildNeighborhoodDense_impl<dim_v, scalar_t>(args...); }, nQuery, functionArguments);
    });
}
